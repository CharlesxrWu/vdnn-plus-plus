#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <cmath>
#include <limits>
#include <cstdlib>

#include <hipblas.h>
#include <hiprand.h>
#include <hipDNN.h>

// #include <opencv2/opencv.hpp>
#include <hip/hip_runtime_api.h>

typedef unsigned char uchar;

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)


// #define checkCudaErrors(status) do {                                   \
//     std::stringstream _error;                                          \
//     if (status != 0) {                                                 \
//       _error << "Cuda failure: " << status;                            \
//       FatalError(_error.str());                                        \
//     }                                                                  \
// } while(0)

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

#define checkCUBLAS(expression)                              \
  {                                                          \
    hipblasStatus_t status = (expression);                    \
    if (status != HIPBLAS_STATUS_SUCCESS) {                   \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << _cudaGetErrorEnum(status) << std::endl;   \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

#define checkCURAND(expression)                               \
  {                                                          \
    hiprandStatus_t status = (expression);                     \
    if (status != HIPRAND_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << _cudaGetErrorEnum(status) << std::endl;   \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }


using namespace std;

int N_train = 60000, N_test = 10000;
int rows = 28, cols = 28, channels = 1;
int BW = 16 * 16;						// Block size for GPU kernel

// void roundUp(int a, int b) {

// }

__global__ void fillValue(float *v, int size, int value) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)
		return;
	v[i] = value;
}

__global__ void softmaxLossBackProp(float *y, float *SO, float *dSO, int batch_size, int output_size, float eps) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= batch_size)
		return;
	int cur_class = static_cast<int>(y[i]);
	dSO[i * output_size + cur_class] = -1 / (SO[i * output_size + cur_class] * batch_size + eps);
}

__global__ void inferClass(float *O, float *IO, int batch_size, int output_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= batch_size)
		return;

	float max = O[i * output_size];
	int index = 0;
	for (int j = 1; j < output_size; j++) {
		if (O[i * output_size + j] > max) {
			max = O[i * output_size + j];
			index = j;
		}
	}
	IO[i] = (float)index;
}

int reverseInt(int n) {
	int bytes = 4;
	unsigned char ch[bytes];
	for (int i = 0; i < bytes; i++) {
		ch[i] = (n >> i * 8) & 255;
	}
	int p = 0;
	for (int i = 0; i < bytes; i++) {
		p += (int) ch[i] << (bytes - i - 1) * 8;
	}
	return p;
}


void readMNIST(vector<vector<uchar> > &train_images, vector<vector<uchar> > &test_images, vector<uchar> &train_labels, vector<uchar> &test_labels) {
	string filename_train_images = "data/train-images.idx3-ubyte";
	string filename_train_labels = "data/train-labels.idx1-ubyte";

	string filename_test_images = "data/t10k-images.idx3-ubyte";
	string filename_test_labels = "data/t10k-labels.idx1-ubyte";

	// read train/test images
	for (int i = 0; i < 2; i++) {
		string filename;
		if (i == 0)
			filename = filename_train_images;
		else
			filename = filename_test_images;

		ifstream f(filename.c_str(), ios::binary);
		if (!f.is_open())
			printf("Cannot read MNIST from %s\n", filename.c_str());

		// read metadata
		int magic_number = 0, n_images = 0, n_rows = 0, n_cols = 0;
		f.read((char *) &magic_number, sizeof(magic_number));
		magic_number = reverseInt(magic_number);
		f.read((char *) &n_images, sizeof(n_images));
		n_images = reverseInt(n_images);
		f.read((char *) &n_rows, sizeof(n_rows));
		n_rows = reverseInt(n_rows);
		f.read((char *) &n_cols, sizeof(n_cols));
		n_cols = reverseInt(n_cols);

		for (int k = 0; k < n_images; k++) {
			vector<uchar> temp;
			temp.reserve(n_rows * n_cols);
			for (int j = 0; j < n_rows * n_cols; j++) {
				uchar t = 0;
				f.read((char *)&t, sizeof(t));
				temp.push_back(t);
			}
			if (i == 0)
				train_images.push_back(temp);
			else
				test_images.push_back(temp);
		}
		f.close();

	}

	// read train/test labels
	for (int i = 0; i < 2; i++) {
		string filename;
		if (i == 0)
			filename = filename_train_labels;
		else
			filename = filename_test_labels;

		ifstream f(filename.c_str(), ios::binary);
		if (!f.is_open())
			printf("Cannot read MNIST from %s\n", filename.c_str());

		// read metadata
		int magic_number = 0, n_labels = 0;
		f.read((char *) &magic_number, sizeof(magic_number));
		magic_number = reverseInt(magic_number);
		f.read((char *) &n_labels, sizeof(n_labels));
		n_labels = reverseInt(n_labels);

		for (int k = 0; k < n_labels; k++) {
			uchar t = 0;
			f.read((char *)&t, sizeof(t));
			if (i == 0)
				train_labels.push_back(t);
			else
				test_labels.push_back(t);
		}

		f.close();

	}


}

void printMatrix(float *M, int r, int c) {
	for (int i = 0; i < r; i++) {
		for (int j = 0; j < c; j++) {
			cout << M[i * c + j] << ' ';
		}
		cout << endl;
	}
	cout << endl;
}

class Context {
public:	
	int batch_size, channels;
	int input_rows, input_cols, output_rows, output_cols;
	float learning_rate;
	float *IO;
	float *y;
	float *onevec;
	float *h_IO;
	int input_size;
	int input_size_fc;
	int hidden_size;
	int output_size;
	int input_feature, output_feature;
	hipblasHandle_t cublasHandle;
	// hipdnnTensorDescriptor_t batchTensor, W1Tensor, b1Tensor, W2Tensor, b2Tensor, HTensor, OTensor;
	hipdnnTensorDescriptor_t HTensor, OTensor;
	hipdnnActivationDescriptor_t Reludesc;
	// hipdnnOpTensorDescriptor_t Adddesc, Muldesc;

	hipdnnHandle_t cudnn_handle;
	hiprandGenerator_t curandgen;

	float *h_W1, *h_W2, *h_b1, *h_b2, *h_SO, *h_y;
	float eps;

	// conv
	hipdnnTensorDescriptor_t input_tensor, output_tensor, bias_tensor, pooling_output_tensor;
	hipdnnFilterDescriptor_t filter_desc;
	hipdnnConvolutionDescriptor_t conv_desc;
	hipdnnActivationDescriptor_t actv_desc;
	hipdnnPoolingDescriptor_t pool_desc;
	hipdnnConvolutionFwdAlgo_t conv_fwd_algo;
	hipdnnConvolutionBwdFilterAlgo_t conv_bwdf_algo;
	size_t workspace_size;
	float *workspace;
	float *conv1O, *conv1OA;
	float *conv1filter, *conv1bias;
	float *dconv1filter, *dconv1bias;
	float *dconv1O, *dconv1OA;
	int filter_height, filter_width;
	
	// vdnn
	int req_algo_count;
	hipdnnConvolutionFwdAlgoPerf_t *conv1fwdperf;
	hipdnnConvolutionBwdFilterAlgoPerf_t *conv1bwdfperf;
	hipdnnConvolutionBwdDataAlgoPerf_t *conv1bwddperf;

	Context(int input_size, int batch_size, int hidden_size, float learning_rate, int output_size, int filter_size) {
		this->batch_size = batch_size;
		this->hidden_size = hidden_size;
		this->output_size = output_size;				// number of classes;
		this->channels = 1;
		
		this->batch_size = batch_size = 128;
		input_rows = 112;
		input_cols = 112;
		input_feature = 128;
		output_rows = 112;
		output_cols = 112;
		output_feature = 128;
		filter_height = filter_size, filter_width = filter_size;
		int pad_h = filter_size / 2, pad_w = filter_size / 2, u = 1, v = 1, dilation_h = 1, dilation_w = 1;
		this->input_size = input_rows * input_cols * input_feature;
		cout << "input_size: " << this->input_size << endl;
		// input_size_fc = output_rows * output_cols * output_feature;
		this->learning_rate = learning_rate;
		eps = 1e-8;
		workspace_size = 0;
		workspace = NULL;

		// find time for conv or pool
		bool conv_test = false;

		checkCUBLAS(hipblasCreate(&cublasHandle));
		checkCUDNN(hipdnnCreate(&cudnn_handle));
		checkCURAND(hiprandCreateGenerator(&curandgen, HIPRAND_RNG_PSEUDO_DEFAULT));

		//vdnn
		req_algo_count = 10;
		conv1fwdperf = (hipdnnConvolutionFwdAlgoPerf_t *)malloc(req_algo_count * sizeof(hipdnnConvolutionFwdAlgoPerf_t));
		conv1bwdfperf = (hipdnnConvolutionBwdFilterAlgoPerf_t *)malloc(req_algo_count * sizeof(hipdnnConvolutionBwdFilterAlgoPerf_t));
		conv1bwddperf = (hipdnnConvolutionBwdDataAlgoPerf_t *)malloc(req_algo_count * sizeof(hipdnnConvolutionBwdDataAlgoPerf_t));

		// conv
		checkCUDNN(hipdnnCreateTensorDescriptor(&input_tensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&output_tensor));
		checkCUDNN(hipdnnCreateTensorDescriptor(&bias_tensor));
		checkCUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
		checkCUDNN(hipdnnSetTensor4dDescriptor(input_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, input_feature, input_rows, input_cols));
		checkCUDNN(hipdnnSetTensor4dDescriptor(output_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, output_feature, output_rows, output_cols));
		checkCUDNN(hipdnnSetTensor4dDescriptor(bias_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_feature, 1, 1));

		checkCUDNN(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
											 output_feature, input_feature, filter_height, filter_width));

		checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
		
		checkCUDNN(hipdnnSetConvolution2dDescriptor(conv_desc, pad_h, pad_w, u, v, dilation_h, dilation_w,
													HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

		checkCUDNN(hipdnnCreateActivationDescriptor(&actv_desc));
		checkCUDNN(hipdnnSetActivationDescriptor(actv_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 1e-8));

		int pooling_output_rows, pooling_output_cols;
		if (!conv_test) {
			checkCUDNN(hipdnnCreatePoolingDescriptor(&pool_desc));
			u = v = 2;
			filter_height = filter_width = 2;
			pad_h = pad_w = 0;
			pooling_output_rows = (input_rows + 2 * pad_w - filter_width) / u, pooling_output_cols = (input_cols + 2 * pad_h - filter_height) / v;
			checkCUDNN(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
													filter_height, filter_width,
													pad_h, pad_w,
													u, v));
			checkCUDNN(hipdnnCreateTensorDescriptor(&pooling_output_tensor));
			checkCUDNN(hipdnnSetTensor4dDescriptor(pooling_output_tensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, input_feature, pooling_output_rows, pooling_output_cols));
		}

		int ret_algo_count;
		int n;
		// cout << "waiting..\n";
		// cin >> n;
		if (conv_test) {
			checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(cudnn_handle, input_tensor, filter_desc, conv_desc, output_tensor,
															 req_algo_count, &ret_algo_count, conv1fwdperf));
			cerr << "Printing forward conv algo perf\n";
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM " << HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM << endl;
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM " << HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM << endl;
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_GEMM " << HIPDNN_CONVOLUTION_FWD_ALGO_GEMM << endl;
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT " << HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT << endl;
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_FFT " << HIPDNN_CONVOLUTION_FWD_ALGO_FFT << endl;
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING " << HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING << endl;
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD " << HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD << endl;
			cerr << "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED " << HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED << endl;
			for (int i = 0; i < ret_algo_count; i++) {
				cerr << i << endl;
				cerr << "algo: " << conv1fwdperf[i].algo << endl;
				cerr << "status: " << hipdnnGetErrorString(conv1fwdperf[i].status) << endl;
				cerr << "time(ms): " << conv1fwdperf[i].time << endl;
				cerr << "memory(bytes): " << conv1fwdperf[i].memory << endl;
				cerr << "mathType: " << conv1fwdperf[i].mathType << endl;
				cerr << endl;
			}
			conv_fwd_algo = conv1fwdperf[0].algo;
			workspace_size = conv1fwdperf[0].memory;
		}

		// {
		// 	int n;
		// 	cout << "waiting..\n";
		// 	cin >> n;
		// }
		float alpha = 1.0, beta = 0.0;

		void *layer_input, *layer_output, *workspace, *W, *b, *pooling_layer_output;
		checkCudaErrors(hipMalloc(&layer_input, batch_size * input_feature * input_rows * input_cols * sizeof(float)));
		checkCudaErrors(hipMalloc(&layer_output, batch_size * output_feature * output_rows * output_cols * sizeof(float)));
		checkCudaErrors(hipMalloc(&W, output_feature * input_feature * filter_height * filter_width * sizeof(float)));
		checkCudaErrors(hipMalloc(&b, 1 * output_feature * 1 * 1 * sizeof(float)));
		checkCudaErrors(hipMalloc(&workspace, workspace_size));
		checkCudaErrors(hipMalloc(&pooling_layer_output, batch_size * input_feature * pooling_output_rows * pooling_output_cols * sizeof(float)));

		int n_iters = 100;
		hipEvent_t start, stop;
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));

		if (!conv_test) {
			std::vector<float> pool_compute_times;
			for (int i = 0; i < n_iters; i++) {
				float milli = 0;
				checkCudaErrors(hipEventRecord(start));
				checkCUDNN(hipdnnPoolingForward(cudnn_handle, pool_desc,
												&alpha,
												input_tensor, layer_input,
												&beta,
												pooling_output_tensor, pooling_layer_output));

				checkCudaErrors(hipEventRecord(stop));
				checkCudaErrors(hipEventSynchronize(stop));
				checkCudaErrors(hipEventElapsedTime(&milli, start, stop));
				pool_compute_times.push_back(milli);
			}
			fstream f_pool_compute;
			f_pool_compute.open("pool_compute_time.dat", ios_base::out);
			for (int i = 0; i < n_iters; i++) {
				f_pool_compute << pool_compute_times[i] << endl;
			}
			f_pool_compute.close();
			exit(0);
		}

		std::vector<float> compute_times;
		for (int i = 0; i < n_iters; i++) {
			float milli = 0;
			checkCudaErrors(hipEventRecord(start));

			checkCUDNN(hipdnnConvolutionForward(cudnn_handle, &alpha, 
												input_tensor, layer_input,
												filter_desc, W,
												conv_desc, conv_fwd_algo,
												workspace, workspace_size,
												&beta,
												output_tensor, layer_output));
			checkCUDNN(hipdnnAddTensor(cudnn_handle, &alpha, 
										bias_tensor, b, 
										&alpha,
										output_tensor, layer_output));

			checkCUDNN(hipdnnActivationForward(cudnn_handle, actv_desc,
												&alpha,
												output_tensor, layer_output,
												&beta,
												output_tensor, layer_output));

			checkCudaErrors(hipEventRecord(stop));
			checkCudaErrors(hipEventSynchronize(stop));
			checkCudaErrors(hipEventElapsedTime(&milli, start, stop));
			compute_times.push_back(milli);
		}

		void *h_layer_input;
		checkCudaErrors(hipHostMalloc(&h_layer_input, batch_size * input_feature * input_rows * input_cols * sizeof(float)));
		std::vector<float> transfer_times;
		for (int i = 0; i < n_iters; i++) {
			float milli;
			checkCudaErrors(hipEventRecord(start));			
			checkCudaErrors(hipMemcpyAsync(h_layer_input, layer_input, batch_size * input_feature * input_rows * input_cols * sizeof(float), hipMemcpyDeviceToHost, NULL));
			checkCudaErrors(hipEventRecord(stop));
			checkCudaErrors(hipEventSynchronize(stop));
			checkCudaErrors(hipEventElapsedTime(&milli, start, stop));
			transfer_times.push_back(milli);
		}

		fstream f_compute;
		fstream f_transfer;
		char filter_char[10];
		sprintf(filter_char, "%d", filter_size);
		std::string compute_filename = "compute_time_";
		std::string transfer_filename = "transfer_time_";
		compute_filename.append(filter_char);
		compute_filename.append(".dat");
		transfer_filename.append(filter_char);
		transfer_filename.append(".dat");
		f_compute.open(compute_filename.c_str(), ios_base::out);
		f_transfer.open(transfer_filename.c_str(), ios_base::out);
		for (int i = 0; i < n_iters; i++) {
			f_compute << compute_times[i] << std::endl;
			f_transfer << transfer_times[i] << std::endl;
		}
		f_compute.close();
		f_transfer.close();

		exit(0);
		

	}

	~Context() {
		checkCUBLAS(hipblasDestroy(cublasHandle));
		checkCURAND(hiprandDestroyGenerator(curandgen));

	}

	void forwardPropagate(bool train=true) {
		
		

		

	}

	

	void train(int num_iter, float *train_images, float *train_labels, float *test_images, float *test_labels, int N) {
		// int image_size = rows * cols * channels;

		for (int iter = 0; iter < num_iter; iter++) {
			int image_id = iter % (N / batch_size);


			this->forwardPropagate();
			

			checkCudaErrors(hipDeviceSynchronize());
			exit(0);
			
			checkCudaErrors(hipDeviceSynchronize());
		}
	}

	int test(float *test_images, float *test_labels, int N) {
		// int image_size = rows * cols * channels;
		int start = 0;
		int size = batch_size;
		int count = 0;
		while (start < N) {
			if (start + size >= N)
				size = N - start;
			// checkCudaErrors(hipMemcpy(X, &test_images[start * input_size], input_size * size * sizeof(float), hipMemcpyHostToDevice));
			// checkCudaErrors(hipMemcpy(y, &test_labels[start], size * sizeof(float), hipMemcpyHostToDevice));
			this->forwardPropagate(false);
			checkCudaErrors(hipDeviceSynchronize());
			for (int i = 0; i < size; i++) {
				if (h_IO[i] == test_labels[start + i])
					count++;
				// cout << h_IO[i] << ' ';
			}
			start = start + size;
		}
		return count;

	}


};

int main(int argc, char *argv[]) {
	float *f_train_images, *f_train_labels, *f_test_images, *f_test_labels;
	int input_size = rows * cols * channels;
	f_train_images = (float *)malloc(N_train * input_size * sizeof(float));
	f_train_labels = (float *)malloc(N_train * sizeof(float));
	f_test_images = (float *)malloc(N_test * input_size * sizeof(float));
	f_test_labels = (float *)malloc(N_test * sizeof(float));
	

	float l_rate = 1e-3;
	int hidden_size = 50;
	int batch_size = 16;
	int output_size = 10;
	int filter_size = 3;
	if (argc >= 2) {
		filter_size = atoi(argv[1]);
	}
	Context context(input_size, batch_size, hidden_size, l_rate, output_size, filter_size);
	int n_iter = 10000;
	int n_rep = 10;


	for (int i = 0; i < n_rep; i++) {
		context.train(n_iter, f_train_images, f_train_labels, f_test_images, f_test_labels, N_train);
		cout << context.test(f_test_images, f_test_labels, N_test) << endl;
	}


}