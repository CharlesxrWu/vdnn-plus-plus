
#include <hip/hip_runtime.h>
#include <iostream>
#include <pthread.h>
#include <cstdlib>
#include <vector>
#include <cmath>
#define NUM_COMPRESSION_THREADS 8
#define COMPRESSION_DISCRETIZATION_FACTOR 8
#define COMPRESSION_BATCH_SIZE 32
#define ALLOC_AND_COMPRESS 0
#define BYTE_SIZE 8

using namespace std;

struct CompressedData {
	void ***data;
	bool **slot_taken;
	unsigned int *mask;
};

struct CompressionMetadata {
	long total_compression_batches;
	long *num_elements, *start_pos, *num_compression_batches;
	long **slot_size; 
};

struct CompressionThreadArgs {
	CompressedData *compressed_data;
	float *original_data;
	CompressionMetadata *compression_metadata;
	int thread_num;
};

struct Position2dArray {
	long slot;
	long offset;
};

long layer_sizes_alexnet[] = {56l * 56 * 96, 28l * 28 * 96, 27l * 27 * 256, 13l * 13 * 256, 13l * 12 * 384, 13l * 12 * 384, 13l * 13 * 256, 6l * 6 * 256};
bool layer_compress_alexnet[] = {false, false, true, false, true, true, true, true};
long layer_density_alexnet[] = {50, 80, 40, 60, 70, 70, 30, 60};
int num_layers_alexnet = 8;

long layer_sizes_vgg[] = {224l * 224 * 64, 
							224l * 224 * 64, 
							112l * 112 * 64, 
							112l * 112 * 128, 
							112l * 112 * 128, 
							56l * 56 * 128, 
							56l * 56 * 256, 
							56l * 56 * 256, 
							56l * 56 * 256, 
							28l * 28 * 256, 
							28l * 28 * 512, 
							28l * 28 * 512, 
							28l * 28 * 512, 
							14l * 14 * 512, 
							14l * 14 * 512, 
							14l * 14 * 512, 
							14l * 14 * 512, 
							7l * 7 * 512};

long layer_density_vgg[] = {50,
							20, 
							30,
							20,
							10,
							20,
							20,
							20,
							10,
							20,
							20,
							10,
							10,
							10,
							20,
							20,
							10,
							15
							};
bool layer_compress_vgg[] = {true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true,
						true};

int num_layers_vgg = 18;

long *layer_sizes = layer_sizes_alexnet;
bool *layer_compress = layer_compress_alexnet;
long *layer_density = layer_density_alexnet;
int num_layers = num_layers_alexnet;

void *compressThread(void *);
void *decompressThread(void *);

int main() {
	int batch_size = 128;
	for (int i = 0; i < num_layers; i++) {
		layer_sizes[i] *= batch_size;
	}
	// allocate space for compressed_data_pointers
	CompressedData *compressed_data = (CompressedData *)malloc(num_layers * sizeof(CompressedData));
	for (int i = 0; i < num_layers; i++) {
		if (layer_compress[i]) {
			hipHostMalloc((void **)&compressed_data[i].data, NUM_COMPRESSION_THREADS * sizeof(void **), hipHostMallocDefault);
			hipHostMalloc((void **)&compressed_data[i].slot_taken, NUM_COMPRESSION_THREADS * sizeof(bool *), hipHostMallocDefault);
			for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
				hipHostMalloc((void **)&compressed_data[i].data[j], COMPRESSION_DISCRETIZATION_FACTOR * sizeof(void *), hipHostMallocDefault);
				hipHostMalloc((void **)&compressed_data[i].slot_taken[j], COMPRESSION_DISCRETIZATION_FACTOR * sizeof(bool), hipHostMallocDefault);
			}
		}
	}

	if (COMPRESSION_BATCH_SIZE != sizeof(unsigned int) * BYTE_SIZE) {
		std::cout << "Panic!! COMPRESSION_BATCH_SIZE = 32\n sizeof(unsigned int) = " << sizeof(unsigned int) << endl;
		std::cout << "Sizes do not match\n";
	}

	CompressionMetadata *compression_metadata = (CompressionMetadata *)malloc(num_layers * sizeof(CompressionMetadata));
	for (int i = 0; i < num_layers; i++) {
		if (layer_compress[i]) {
			compression_metadata[i].num_compression_batches = (long *)malloc(NUM_COMPRESSION_THREADS * sizeof(long));
			compression_metadata[i].num_elements = (long *)malloc(NUM_COMPRESSION_THREADS * sizeof(long));
			compression_metadata[i].start_pos = (long *)malloc(NUM_COMPRESSION_THREADS * sizeof(long));
			compression_metadata[i].slot_size = (long **)malloc(NUM_COMPRESSION_THREADS * sizeof(long *));
			for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
				compression_metadata[i].slot_size[j] = (long *)malloc(COMPRESSION_DISCRETIZATION_FACTOR * sizeof(long));
			}
			compression_metadata[i].total_compression_batches = ceil(1.0 * layer_sizes[i] / COMPRESSION_BATCH_SIZE);
			for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
				compression_metadata[i].num_compression_batches[j] = compression_metadata[i].total_compression_batches / NUM_COMPRESSION_THREADS;
			}
			long num_leftout_compression_batches = compression_metadata[i].total_compression_batches % NUM_COMPRESSION_THREADS;
			for (int j = 0; j < num_leftout_compression_batches; j++) {
				compression_metadata[i].num_compression_batches[NUM_COMPRESSION_THREADS - 1 - j] += 1;
			}
			for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
				if (j < NUM_COMPRESSION_THREADS - 1) {
					compression_metadata[i].num_elements[j] = compression_metadata[i].num_compression_batches[j] * COMPRESSION_BATCH_SIZE;
				}
				else {
					compression_metadata[i].num_elements[j] = (compression_metadata[i].num_compression_batches[j] - 1) * COMPRESSION_BATCH_SIZE;
					if (layer_sizes[i] % COMPRESSION_BATCH_SIZE == 0)
						compression_metadata[i].num_elements[j] += COMPRESSION_BATCH_SIZE;
					else
						compression_metadata[i].num_elements[j] += layer_sizes[i] % COMPRESSION_BATCH_SIZE;
				}
			}
			long cumulative_start_pos = 0;
			for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
				compression_metadata[i].start_pos[j] = cumulative_start_pos;
				cumulative_start_pos += compression_metadata[i].num_elements[j];
			}
			for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
				for (int k = 0; k < COMPRESSION_DISCRETIZATION_FACTOR; k++) {
					compression_metadata[i].slot_size[j][k] = compression_metadata[i].num_elements[j] / COMPRESSION_DISCRETIZATION_FACTOR;
				}
				for (int k = 0; k < compression_metadata[i].num_elements[j] % COMPRESSION_DISCRETIZATION_FACTOR; k++) {
					compression_metadata[i].slot_size[j][k] += 1;
				}
			}
		}
	}

	float **h_layer_input = (float **)malloc(num_layers * sizeof(float *));
	// create args for compression threads
	CompressionThreadArgs **compression_thread_args = (CompressionThreadArgs **)malloc(num_layers * sizeof(CompressionThreadArgs *));
	for (int i = 0; i < num_layers; i++) {
		compression_thread_args[i] = (CompressionThreadArgs *)malloc(NUM_COMPRESSION_THREADS * sizeof(CompressionThreadArgs));
		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			compression_thread_args[i][j].compressed_data = &compressed_data[i];
			compression_thread_args[i][j].original_data = h_layer_input[i];
			compression_thread_args[i][j].compression_metadata = &compression_metadata[i];
			compression_thread_args[i][j].thread_num = j;
		}
	}
	

	vector<float> compression_times, decompression_times;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	for (int i = 0; i < num_layers; i++) {
		if (!layer_compress[i]) {
			compression_times.push_back(0);
			continue;
		}

		float milli;
		hipHostMalloc((void **)&h_layer_input[i], layer_sizes[i] * sizeof(float), hipHostMallocDefault);
		pthread_t threads[NUM_COMPRESSION_THREADS];

		// generate data
		for (long j = 0; j < layer_sizes[i]; j++) {
			if (rand() % 100 < layer_density[i])
				h_layer_input[i][j] = 1;
			else
				h_layer_input[i][j] = 0;
		}

		cout << "starting " << i << endl;
		hipEventRecord(start);

		hipHostMalloc(&compressed_data[i].mask, ceil(1.0 * layer_sizes[i] / COMPRESSION_BATCH_SIZE) * sizeof(unsigned int), hipHostMallocDefault);
		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			compression_thread_args[i][j].original_data = h_layer_input[i];
		}

		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			pthread_create(&threads[j], NULL, &compressThread, (void *)&compression_thread_args[i][j]);
		}
		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			pthread_join(threads[j], NULL);
		}

		hipHostFree(h_layer_input[i]);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		compression_times.push_back(milli);
	}

	size_t total_size = 0;
	size_t total_size_uncompressed = 0;
	for (int i = 0; i < num_layers; i++) {
		total_size_uncompressed += layer_sizes[i];
		if (!layer_compress[i]) {
			total_size += layer_sizes[i];
			continue;
		}
		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			for (int k = 0; k < COMPRESSION_DISCRETIZATION_FACTOR; k++) {
				if (compressed_data[i].slot_taken[j][k]) {
					total_size += compression_metadata[i].slot_size[j][k];
				}
			}
		}
	}
	std::cout << "total_size_compressed(MB): " << total_size * sizeof(float) / (1.0 * 1024 * 1024) << std::endl;
	std::cout << "total_size_uncompressed(MB): " << total_size_uncompressed * sizeof(float) / (1.0 * 1024 * 1024) << std::endl;


	// decompression
	for (int i = 0; i < num_layers; i++) {
		float milli;
		if (!layer_compress[i]) {
			decompression_times.push_back(0);
			continue;
		}

		pthread_t threads[NUM_COMPRESSION_THREADS];
		hipEventRecord(start);
		hipHostMalloc(&h_layer_input[i], layer_sizes[i] * sizeof(float), hipHostMallocDefault);

		cout << "starting " << i << endl;



		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			compression_thread_args[i][j].original_data = h_layer_input[i];
		}

		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			pthread_create(&threads[j], NULL, &decompressThread, (void *)&compression_thread_args[i][j]);
		}
		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			pthread_join(threads[j], NULL);
		}

		for (int j = 0; j < NUM_COMPRESSION_THREADS; j++) {
			for (int k = 0; k < COMPRESSION_DISCRETIZATION_FACTOR; k++) {
				if (compressed_data[i].slot_taken[j][k]) {
					hipHostFree(compressed_data[i].data[j][k]);
				}
			}
		}

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&milli, start, stop);
		decompression_times.push_back(milli);
	}

	float total_compression_time = 0, total_decompression_time = 0;
	for (int i = 0; i < num_layers; i++) {
		cout << i << " " << compression_times[i] << endl;
		total_compression_time += compression_times[i];
	}
	cout << endl;
	for (int i = 0; i < num_layers; i++) {
		cout << i << " " << decompression_times[i] << endl;
		total_decompression_time += decompression_times[i];
	}
	cout << "total compression time(ms): " << total_compression_time << endl;
	cout << "total decompression time(ms): " << total_decompression_time << endl;

}

void *compressThread(void *arg) {
	CompressionThreadArgs *args = (CompressionThreadArgs *)arg;
	
	// retrieve args
	float *original_data = (float *)(args->original_data);
	CompressedData compressed_data = *(args->compressed_data);
	CompressionMetadata compression_metadata = *(args->compression_metadata);
	int thread_num = args->thread_num;

	// reset compressed_data.slot_taken
	for (int i = 0; i < COMPRESSION_DISCRETIZATION_FACTOR; i++) {
		compressed_data.slot_taken[thread_num][i] = false;
	}

	Position2dArray compressed_data_pos, mask_pos;
	compressed_data_pos.slot = -1, compressed_data_pos.offset = -1;
	mask_pos.slot = compression_metadata.start_pos[thread_num] / COMPRESSION_BATCH_SIZE;
	mask_pos.offset = 0;

	for (long i = compression_metadata.start_pos[thread_num]; i < compression_metadata.start_pos[thread_num] + compression_metadata.num_elements[thread_num]; i++) {
		if (mask_pos.offset == 0)
			compressed_data.mask[mask_pos.slot] = 0;

		if (original_data[i] > 0) {
			if (compressed_data_pos.offset == -1 or compressed_data_pos.offset == compression_metadata.slot_size[thread_num][compressed_data_pos.slot]) {
				compressed_data_pos.slot += 1;
#ifdef ALLOC_AND_COMPRESS
				hipHostMalloc((void **)&compressed_data.data[thread_num][compressed_data_pos.slot], compression_metadata.slot_size[thread_num][compressed_data_pos.slot] * sizeof(float), hipHostMallocDefault);
#endif
				compressed_data.slot_taken[thread_num][compressed_data_pos.slot] = true;
				compressed_data_pos.offset = 0;
			}
			compressed_data.mask[mask_pos.slot] = (compressed_data.mask[mask_pos.slot] << 1) + 1;
			((float ***)compressed_data.data)[thread_num][compressed_data_pos.slot][compressed_data_pos.offset] = original_data[i];
			compressed_data_pos.offset += 1;
		}
		else {
			compressed_data.mask[mask_pos.slot] = (compressed_data.mask[mask_pos.slot] << 1);
		}
		mask_pos.offset += 1;
		if (mask_pos.offset == COMPRESSION_BATCH_SIZE) {
			mask_pos.slot += 1;
			mask_pos.offset = 0;
		}

	}
}

void *decompressThread(void *arg) {
	CompressionThreadArgs *args = (CompressionThreadArgs *)arg;
	
	// retrieve args
	float *original_data = (float *)(args->original_data);
	CompressedData compressed_data = *(args->compressed_data);
	CompressionMetadata compression_metadata = *(args->compression_metadata);
	int thread_num = args->thread_num;

	Position2dArray compressed_data_pos, mask_pos;
	compressed_data_pos.slot = 0, compressed_data_pos.offset = 0;
	mask_pos.slot = compression_metadata.start_pos[thread_num] / COMPRESSION_BATCH_SIZE;
	mask_pos.offset = 0;

	// handling the last part where it might not be completely filled
	if (thread_num == NUM_COMPRESSION_THREADS - 1) {
		if (compression_metadata.num_elements[thread_num] % COMPRESSION_BATCH_SIZE != 0) {
			compressed_data.mask[mask_pos.slot + compression_metadata.num_elements[thread_num] / COMPRESSION_BATCH_SIZE] = compressed_data.mask[mask_pos.slot + compression_metadata.num_elements[thread_num] / COMPRESSION_BATCH_SIZE] << (COMPRESSION_BATCH_SIZE - compression_metadata.num_elements[thread_num] % COMPRESSION_BATCH_SIZE);
		}
	}

	for (long i = compression_metadata.start_pos[thread_num]; i < compression_metadata.start_pos[thread_num] + compression_metadata.num_elements[thread_num]; i++) {

		if (compressed_data.mask[mask_pos.slot] & 0x80000000 > 0) {
			original_data[i] = ((float ***)compressed_data.data)[thread_num][compressed_data_pos.slot][compressed_data_pos.offset];
			compressed_data_pos.offset += 1;
			if (compressed_data_pos.offset == compression_metadata.slot_size[thread_num][compressed_data_pos.slot]) {
				compressed_data_pos.slot += 1;
				compressed_data_pos.offset = 0;
			}
		}
		else {
			original_data[i] = 0;
		}
		compressed_data.mask[mask_pos.slot] = (compressed_data.mask[mask_pos.slot] << 1);
		mask_pos.offset += 1;
		if (mask_pos.offset == COMPRESSION_BATCH_SIZE) {
			mask_pos.slot += 1;
			mask_pos.offset = 0;
		}

	}
}